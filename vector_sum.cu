#define N 1200
#define THREADS 1024

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__global__ void vecAdd(int *a, int *b, int *c);

int main(){
    int *a, *b, *c;
    int *dev_a, *dev_b, *dev_c;
    int size;

    size = N*sizeof(int);

    hipMalloc((void**) &dev_a, size);
    hipMalloc((void**) &dev_b, size);
    hipMalloc((void**) &dev_c, size);

    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    for(int i = 0; i < N; i++){
        a[i] = b[i] = i;
        c[i] = 0;
    }

    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

    vecAdd<<<(int)ceil(THREADS/N),N>>>(dev_a, dev_b, dev_c);

    hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

    for(int i = 0; i < N; i++){
        printf("c[%d] = %d\n", i, c[i]);
    }

    free(a);
    free(b);
    free(c);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    
    exit(0);
}

__global__ void vecAdd(int *a, int *b, int *c){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i < N){
        c[i] = a[i] + b[i];
        printf("Sou a thread %d em %d\n", threadIdx.x, i);
    }
}