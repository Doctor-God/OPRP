
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ const char *str = "Hello World!";
const char str_length = 12;

__global__ void hello(){
    printf("%c\n", str[threadIdx.x % str_length]);
}

int main(void){
    // int num_threads = str_length;
    // int num_blocks = 2;
    // dim3 dimBlock(16, 16);
    // dim3 dimGrid(32, 32);
    hello<<<1,str_length>>>();
    hipDeviceSynchronize();
    
    return 0;
}